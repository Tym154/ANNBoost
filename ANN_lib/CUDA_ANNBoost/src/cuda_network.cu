#include "hip/hip_runtime.h"
#include "../include/cuda_network.hpp"
#include "hip/hip_runtime.h"

void network::network_forward_propagation_GPU(const std::vector<double> &input){
    for(size_t i = 0; i < layers[0].nodes_in_layer.size(); i++){
        layers[0].nodes_in_layer[i].activation = input[i];
    }

    for(size_t i = 1; i < layers.size(); i++){
        layers[i].layer_forward_propagation_GPU(layers[i-1].nodes_in_layer);
    }
}

void network::network_backward_propagation_GPU(const std::vector<double> &expected_activations){
    network_calculate_output_losses_GPU(expected_activations);

    std::vector<double> losses = layers.back().layer_backward_propagation_GPU(output_layer_losses, layers[layers.size() - 2].nodes_in_layer, learning_rate);

    for(int i = layers.size()-2; i > 0; i--){
        losses = layers[i].layer_backward_propagation_GPU(losses, layers[i-1].nodes_in_layer, learning_rate);
    }
}

void network::network_calculate_output_losses_GPU(const std::vector<double> &expected_activations){
    output_layer_losses.assign(output_layer_losses.size(), 0.0);

    int num_nodes = layers[layers.size() - 1].nodes_in_layer.size();

    std::vector<double> host_expected(num_nodes);
    std::vector<double> host_activation(num_nodes);
    std::vector<double> host_output(num_nodes);
    float host_latest_network_cost = 0.0;

    for(int i = 0; i < num_nodes; i++){
        host_expected[i] = expected_activations[i];
        host_activation[i] = layers[layers.size() - 1].nodes_in_layer[i].activation;
    }

    size_t expected_size = num_nodes * sizeof(double);
    size_t activation_size = num_nodes * sizeof(double);
    size_t output_size = num_nodes * sizeof(double);

    double *d_expected_activations, *d_activations, *d_outputs;
    float *d_latest_net_cost;

    hipMalloc(&d_expected_activations, expected_size);
    hipMalloc(&d_activations, activation_size);
    hipMalloc(&d_outputs, output_size);
    hipMalloc(&d_latest_net_cost, sizeof(float));

    hipMemcpy(d_expected_activations, host_expected.data(), expected_size, hipMemcpyHostToDevice);
    hipMemcpy(d_activations, host_activation.data(), activation_size, hipMemcpyHostToDevice);
    hipMemcpy(d_outputs, host_output.data(), output_size, hipMemcpyHostToDevice);
    hipMemcpy(d_latest_net_cost, &host_latest_network_cost, sizeof(float), hipMemcpyHostToDevice);


    int threads_per_block = 512;
    int num_blocks = (num_nodes + threads_per_block - 1) / threads_per_block;

    latest_network_cost = 0;
    calculate_output_losses<<<num_blocks, threads_per_block>>>(d_outputs, d_expected_activations, d_activations, num_nodes, d_latest_net_cost);

    hipDeviceSynchronize();

    hipMemcpy(&host_latest_network_cost, d_latest_net_cost, sizeof(float), hipMemcpyDeviceToHost);
    latest_network_cost = double(host_latest_network_cost);

    hipMemcpy(host_output.data(), d_outputs, output_size, hipMemcpyDeviceToHost);
    for(size_t i = 0; i < output_layer_losses.size(); i++){
        output_layer_losses[i] += host_output[i];
    }

    hipFree(d_expected_activations);
    hipFree(d_activations);
    hipFree(d_outputs);
    hipFree(d_latest_net_cost);
}

__global__ void calculate_output_losses(double* d_outputs, double* d_expected_activations, double* d_activations, int num_nodes, float* latest_net_cost){
    int node_index = blockIdx.x * blockDim.x + threadIdx.x;

    if(node_index < num_nodes){
        d_outputs[node_index] = d_expected_activations[node_index] - d_activations[node_index];
        // Works only for GPU's with compute capability of 6 and higher
        atomicAdd(latest_net_cost, fabs(d_outputs[node_index])); 
    }
}